#include "hip/hip_runtime.h"
#include "life.h"
#include "util.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#define SWAP_BOARDS( b1, b2 )  do { \
  char* temp = b1; \
  b1 = b2; \
  b2 = temp; \
} while(0)

#define BOARD( __board, __i, __j )  (__board[(__i) + LDA*(__j)])

// __global__ void check(char *isalive, char *outboard, char *inboard, int nrows, int ncols, int i, int j, int (*mod)(int, int), char (*alivep)(char, char)) {
//                     const int LDA = nrows;
//                     const int inorth = mod (i-1, nrows);
//                     const int isouth = mod (i+1, nrows);
//                     const int jwest = mod (j-1, ncols);
//                     const int jeast = mod (j+1, ncols);

//                     const char neighbor_count = 
//                         BOARD (inboard, inorth, jwest) + 
//                         BOARD (inboard, inorth, j) + 
//                         BOARD (inboard, inorth, jeast) + 
//                         BOARD (inboard, i, jwest) +
//                         BOARD (inboard, i, jeast) + 
//                         BOARD (inboard, isouth, jwest) +
//                         BOARD (inboard, isouth, j) + 
//                         BOARD (inboard, isouth, jeast);

//                     BOARD(outboard, i, j) = alivep (neighbor_count, BOARD (inboard, i, j));
// }

__device__ int d_mod_v2 (int x, int m)
{
  return (x < 0) ? ((x % m) + m) : (x % m);
}

__device__ char d_alivep_v2 (char count, char state)
{
  return (! state && (count == (char) 3)) ||
    (state && (count >= 2) && (count <= 3));
}

__global__ void GPUInnerLoop_v2(char *outboard, char *inboard, int nrows, int ncols) 
{
   //calculates unique thread ID in the block
  //  int t= (blockDim.x*blockDim.y)*threadIdx.z+    (threadIdx.y*blockDim.x)+(threadIdx.x); 
   
  //  //calculates unique block ID in the grid
  //  int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
   
  //  //block size (this is redundant though)
  //  int T= blockDim.x*blockDim.y*blockDim.z;
   
  //  //grid size (this is redundant though)
  //  int B= gridDim.x*gridDim.y*gridDim.z;
   
   
   /*
   * Each cell in the matrix is assigned to a different thread.
   * Each thread do O(number of asssigned cell) computation.
   * Assigned cells of different threads does not overlape with
   * each other. And so no need for synchronization.
   */
  //printf("thread Idx x y z is:\t%d\t%d\t%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
  //printf("block dimision x y z is:%d\t%d\t%d\n", blockDim.x, blockDim.y, blockDim.z);
  for (int i = threadIdx.x; i < nrows; i += blockDim.x)
  {
    for (int j = threadIdx.y; j < ncols; j+= blockDim.y)
    {
      
        //revise mod and alivep
                const int LDA = nrows;
                
                const int inorth = d_mod_v2 (i-1, nrows);
                const int isouth = d_mod_v2 (i+1, nrows);
                const int jwest = d_mod_v2 (j-1, ncols);
                const int jeast = d_mod_v2 (j+1, ncols);

                const char neighbor_count = 
                    BOARD (inboard, inorth, jwest) + 
                    BOARD (inboard, inorth, j) + 
                    BOARD (inboard, inorth, jeast) + 
                    BOARD (inboard, i, jwest) +
                    BOARD (inboard, i, jeast) + 
                    BOARD (inboard, isouth, jwest) +
                    BOARD (inboard, isouth, j) + 
                    BOARD (inboard, isouth, jeast);
                //printf("%c\n", d_alivep (neighbor_count, BOARD (inboard, i, j)));
                BOARD(outboard, i, j) = d_alivep_v2 (neighbor_count, BOARD (inboard, i, j));

    }
  }
}


char* cuda_v2_game_of_life (
    char* outboard, 
    char* inboard,
    const int nrows,
    const int ncols,
    const int gens_max,
    const int version,
    const int num_threads,
    const int num_blocks) 
{
    const int LDA = nrows;
    int curgen, i, j;
    printf("current version is: %d\n", version);
    char * d_inboard;
    char * d_outboard;
    hipMalloc((void**)&d_inboard, sizeof(char) * nrows * ncols);
    hipMalloc((void**)&d_outboard, sizeof(char) * nrows * ncols);
    hipMemcpy(d_inboard, inboard, sizeof(char) * nrows * ncols, hipMemcpyHostToDevice);
    for (curgen = 0; curgen < gens_max; curgen++)
    {
        GPUInnerLoop_v2<<<dim3(1,1,1), dim3(32, 32,1)>>>(d_outboard, d_inboard, nrows, ncols);
        hipDeviceSynchronize();
        SWAP_BOARDS( d_outboard, d_inboard );
    }
    hipMemcpy(inboard, d_inboard, sizeof(char) * nrows * ncols, hipMemcpyDeviceToHost);
    hipFree(d_inboard);
    hipFree(d_outboard);
 
    
    /* 
    * We return the output board, so that we know which one contains
    * the final result (because we've been swapping boards around).
    * Just be careful when you free() the two boards, so that you don't
    * free the same one twice!!! 
    */
    return inboard;
}
