#include "hip/hip_runtime.h"
#include "life.h"
#include "util.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#define SWAP_BOARDS( b1, b2 )  do { \
  char* temp = b1; \
  b1 = b2; \
  b2 = temp; \
} while(0)

#define BOARD( __board, __i, __j )  (__board[(__i) + LDA*(__j)])

__device__ int d_mod_v3 (int x, int m)
{
  return (x < 0) ? ((x % m) + m) : (x % m);
}

__device__ char d_alivep_v3 (char count, char state)
{
  return (! state && (count == (char) 3)) ||
    (state && (count >= 2) && (count <= 3));
}

__global__ void GPUInnerLoop_v3(char *outboard, char *inboard, int nrows, int ncols) 
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
 
  if (i < nrows && j < ncols) {
    const int inorth = d_mod_v3 (i-1, nrows);
    const int isouth = d_mod_v3 (i+1, nrows);
    const int jwest = d_mod_v3 (j-1, ncols);
    const int jeast = d_mod_v3 (j+1, ncols);
    const int LDA = nrows;
    const char neighbor_count = 
        BOARD (inboard, inorth, jwest) + 
        BOARD (inboard, inorth, j) + 
        BOARD (inboard, inorth, jeast) + 
        BOARD (inboard, i, jwest) +
        BOARD (inboard, i, jeast) + 
        BOARD (inboard, isouth, jwest) +
        BOARD (inboard, isouth, j) + 
        BOARD (inboard, isouth, jeast);
    //printf("%c\n", d_alivep (neighbor_count, BOARD (inboard, i, j)));
    BOARD(outboard, i, j) = d_alivep_v3 (neighbor_count, BOARD (inboard, i, j));
  }
}

char* cuda_v3_game_of_life (
    char* outboard, 
    char* inboard,
    const int nrows,
    const int ncols,
    const int gens_max,
    const int version,
    const int num_threads,
    const int num_blocks) 
{
    const int LDA = nrows;
    int curgen, i, j;
    printf("current version is: %d\n", version);
    char * d_inboard;
    char * d_outboard;
    hipMalloc((void**)&d_inboard, sizeof(char) * nrows * ncols);
    hipMalloc((void**)&d_outboard, sizeof(char) * nrows * ncols);
    hipMemcpy(d_inboard, inboard, sizeof(char) * nrows * ncols, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(nrows / threadsPerBlock.x + 1, ncols / threadsPerBlock.y + 1);
    for (curgen = 0; curgen < gens_max; curgen++)
    {
        GPUInnerLoop_v3<<<numBlocks, threadsPerBlock>>>(d_outboard, d_inboard, nrows, ncols);
        hipDeviceSynchronize();
        SWAP_BOARDS( d_outboard, d_inboard );
    }
    hipMemcpy(inboard, d_inboard, sizeof(char) * nrows * ncols, hipMemcpyDeviceToHost);
    hipFree(d_inboard);
	  hipFree(d_outboard); 
 
    
    /* 
    * We return the output board, so that we know which one contains
    * the final result (because we've been swapping boards around).
    * Just be careful when you free() the two boards, so that you don't
    * free the same one twice!!! 
    */
    return inboard;
}